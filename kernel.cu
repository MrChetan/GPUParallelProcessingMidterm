#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <cstdlib>
#include <stdlib.h>
#include <ctime>
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include "math.h"
#include <time.h>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <cstdlib>

#define len(x) ((int)log10(x)+1)
#define GRID_X (1u << 12)
#define GRID_Y 1
#define BLOCK_X (1u << 10)
#define BLOCK_Y 1

/* Node of the huffman tree */
struct node {
    int value;
    char letter;
    struct node* left, * right;
};

typedef struct node Node;

/* 81 = 8.1%, 128 = 12.8% and so on. The 27th frequency is the space. Source is Wikipedia */
int englishLetterFrequencies[27] = { 81, 15, 28, 43, 128, 23, 20, 61, 71, 2, 1, 40, 24, 69, 76, 20, 1, 61, 64, 91, 28, 10, 24, 1, 20, 1, 130 };

/*finds and returns the small sub-tree in the forrest*/
int findSmaller(Node* array[], int differentFrom) {
    int smaller;
    int i = 0;

    while (array[i]->value == -1)
        i++;
    smaller = i;
    if (i == differentFrom) {
        i++;
        while (array[i]->value == -1)
            i++;
        smaller = i;
    }

    for (i = 1; i < 27; i++) {
        if (array[i]->value == -1)
            continue;
        if (i == differentFrom)
            continue;
        if (array[i]->value < array[smaller]->value)
            smaller = i;
    }

    return smaller;
}

/*builds the huffman tree and returns its address by reference*/
void buildHuffmanTree(Node** tree) {
    Node* temp;
    Node* array[27];
    int i, subTrees = 27;
    int smallOne, smallTwo;

    for (i = 0; i < 27; i++) {
        array[i] = (Node*)malloc(sizeof(Node));
        array[i]->value = englishLetterFrequencies[i];
        array[i]->letter = i;
        array[i]->left = NULL;
        array[i]->right = NULL;
    }

    while (subTrees > 1) {
        smallOne = findSmaller(array, -1);
        smallTwo = findSmaller(array, smallOne);
        temp = array[smallOne];
        array[smallOne] = (Node*)malloc(sizeof(Node));
        array[smallOne]->value = temp->value + array[smallTwo]->value;
        array[smallOne]->letter = 127;
        array[smallOne]->left = array[smallTwo];
        array[smallOne]->right = temp;
        array[smallTwo]->value = -1;
        subTrees--;
    }

    *tree = array[smallOne];

    return;
}

/* builds the table with the bits for each letter. 1 stands for binary 0 and 2 for binary 1 (used to facilitate arithmetic)*/
void fillTable(int codeTable[], Node* tree, int Code) {
    if (tree->letter < 27)
        codeTable[(int)tree->letter] = Code;
    else {
        fillTable(codeTable, tree->left, Code * 10 + 1);
        fillTable(codeTable, tree->right, Code * 10 + 2);
    }

    return;
}

/*function to compress the input*/
void compressFile(FILE* input, FILE* output, int codeTable[]) {
    char bit, c, x = 0;
    int n, length, bitsLeft = 8;
    int originalBits = 0, compressedBits = 0;

   

    while ((c = fgetc(input)) != 10) {
        originalBits++;
        if (c == 32) {
            length = len(codeTable[26]);
            n = codeTable[26];
        }
        else {
            length = len(codeTable[c - 97]);
            n = codeTable[c - 97];
        }

        while (length > 0) {
            compressedBits++;
            bit = n % 10 - 1;
            n /= 10;
            x = x | bit;
            bitsLeft--;
            length--;
            if (bitsLeft == 0) {
                fputc(x, output);
                x = 0;
                bitsLeft = 8;
            }
            x = x << 1;
        }
    }

    if (bitsLeft != 8) {
        x = x << (bitsLeft - 1);
        fputc(x, output);
    }

    /*print details of compression on the screen*/
    fprintf(stderr, "Original bits = %dn", originalBits * 8);
    fprintf(stderr, "Compressed bits = %dn", compressedBits);
    fprintf(stderr, "Saved %.2f%% of memoryn", ((float)compressedBits / (originalBits * 8)) * 100);

    return;
}


__global__ void compress_file_cuda(char* input, char* output, int codeTable[], int input_length) {
    char bit, c, x = 0;
    int n, length, bitsLeft = 8;
    int originalBits = 0, compressedBits = 0;
    int counter = 0;

    size_t idx = blockDim.x * blockIdx.x + threadIdx.x;
    //printf("chetan text %d \n", idx);

    for (int i = 0; i < idx; i++) {
        originalBits++;
        if (input[i] == ' ') {
            length = ((int)log10((double)codeTable[26]) + 1);
            n = codeTable[26];
            printf("%d length \n", length);
            printf("%d N \n", n);
        }
        else {
            length = ((int)log10((double)codeTable[input[i] - 97]) + 1);
            n = codeTable[input[i] - 97];
            printf("%d length \n", length);
            printf("%d N \n", n);
        }

        while (length > 0) {
            compressedBits++;
            bit = n % 10 - 1;
            n /= 10;
            x = x | bit;
            bitsLeft--;
            length--;
            if (bitsLeft == 0) {
                 output[counter] = x;
                 counter++;
                x = 0;
                bitsLeft = 8;
            }
            x = x << 1;
        }

        i++;
    }

    if (bitsLeft != 8) {
        x = x << (bitsLeft - 1);
        output[counter] = x;
    }

    printf("Original bits = %dn", originalBits * 8);
    printf("Compressed bits = %dn", compressedBits);
    printf("Saved %.2f%% of memoryn", ((float)compressedBits / (originalBits * 8)) * 100);

    return;
}

/*invert the codes in codeTable2 so they can be used with mod operator by compressFile function*/
void invertCodes(int codeTable[], int codeTable2[]) {
    int i, n, copy;

    for (i = 0; i < 27; i++) {
        n = codeTable[i];
        copy = 0;
        while (n > 0) {
            copy = copy * 10 + n % 10;
            n /= 10;
        }
        codeTable2[i] = copy;
    }

    return;
}

int main() {
    Node* tree;
    int codeTable[27], codeTable2[27];
    int codeTable2_GPU[27];
    int compress;
    char filename[20];
    FILE* input, * output;
    FILE* input_gpu;
    char output_gpu[100];

    buildHuffmanTree(&tree);

    fillTable(codeTable, tree, 0);

    invertCodes(codeTable, codeTable2);

    /*get input details from user*/
    printf("Type the name of the file to process:");
    scanf("%s", filename);
    printf("Type 1 to compress and 2 to decompress:");
    scanf("%d", &compress);

    input = fopen(filename, "r");
    output = fopen("output.txt", "w");

    char str[16];
    char str_gpu[16];
    //char output[100];

    int i = 0;
    char a;
    while ((a = fgetc(input)) != 10) {
        printf("chetan %c \n", a);
        str[i] = a;
        i++;
    }

    if (compress == 1) {
        //compressFile(input, output, codeTable2);
        
        dim3 grid(GRID_X, GRID_Y);
        dim3 block(BLOCK_X, BLOCK_Y, 1);

        hipMalloc(&input_gpu, sizeof(input_gpu));
        hipMalloc(&input_gpu, sizeof(input_gpu));
        char* output_gpu;
        // malloc() allocate the memory for n chars
        output_gpu = (char*)malloc(16 * sizeof(char));
        hipMalloc((void**)&codeTable2_GPU, sizeof(int));
        hipMemcpy(codeTable2_GPU, codeTable2, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(&input_gpu, &input, sizeof(input_gpu), hipMemcpyHostToDevice);


        //hipMalloc(&str_gpu, 16 * sizeof(char));
        hipMemcpy(str_gpu, str, sizeof(char), hipMemcpyDeviceToHost);
        compress_file_cuda << <1, 16 >> > (str_gpu, output_gpu, codeTable2_GPU, 15);
        hipDeviceSynchronize();

        hipMemcpy(codeTable2_GPU, codeTable2, sizeof(int), hipMemcpyHostToDevice);

        char* output_array[100];
        hipMemcpy((void*)output_array, (void*)output_gpu, 16 * sizeof(char), hipMemcpyDeviceToHost);

      fwrite(output_array, sizeof(char), sizeof(output_array), output);
      fclose(output);


    }
 

    return 0;
}